#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <cstdio>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>

// 用于判断是否支持CUDA加速
__global__ void test_kernel(int *data, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        data[idx] = idx * 2;
    }
}

// 用于判断是否支持CUDA加速
extern "C" int test_cuda_()
{
    int device_count = 0;
    hipError_t err = hipGetDeviceCount(&device_count);

    if (err != hipSuccess || device_count == 0)
    {
        return 0;
    }

    int *d_data;
    const int size = 1024;
    err = hipMalloc(&d_data, size * sizeof(int));
    if (err != hipSuccess)
    {
        return 0;
    }

    dim3 block(256);
    dim3 grid((size + block.x - 1) / block.x);
    test_kernel<<<grid, block>>>(d_data, size);

    err = hipDeviceSynchronize();
    hipFree(d_data);

    return (err == hipSuccess) ? 1 : 0;
}